#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

//convience func for checking runtime CUDA errors
inline
hipError_t checkCuda(hipError_t result)
{
#if defined (DEBUG) || defined (_DEBUG)
	if(result != hipSuccess)
	{
		fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
		assert(result == hipSuccess);
	}
#endif
	return result;
}

void profileCopy(float* h_a, float* h_b, float* d, unsigned int n, char* descr)
{
	unsigned int bytes = n * sizeof(float);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);
	hipMemcpy(d, h_a, bytes, hipMemcpyHostToDevice);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float milSecs;
	hipEventElapsedTime(&milSecs, start, stop);
	printf("%s Transfers:\n", descr);
	printf("  Host to Device Bandwidth[GB/s]: %f\n", 1e-6*bytes/milSecs);

	hipEventRecord(start, 0);
	hipMemcpy(h_b, d, bytes, hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milSecs, start, stop);
	printf("  Device to Host Bandwidth[GB/s]: %f\n", 1e-6*bytes/milSecs);

	unsigned int i;
	float maxErr = 0.0;
	for(i = 0; i < n; ++i)
		maxErr = max(maxErr, abs(h_a[i]-h_b[i]));
	printf("Maximum Error: %f\n", maxErr);

	hipEventDestroy(start);
	hipEventDestroy(stop);
}

int main(int argc, char** argv)
{
	unsigned int numElem = 4*1024*1024;
	unsigned int byteSize = numElem * sizeof(float);
	float *h_aPageable, *h_bPageable;
	float *h_aPinned, *h_bPinned;
	float* d;

	h_aPageable = (float*)malloc(byteSize);
	h_bPageable = (float*)malloc(byteSize);
	checkCuda(hipHostMalloc((void**)&h_aPinned, byteSize));
	checkCuda(hipHostMalloc((void**)&h_bPinned, byteSize));
	hipMalloc((void**)&d, byteSize);

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	printf("Device: %s\n", prop.name);
	printf("Transfer Size: %dB\n", byteSize);
	profileCopy(h_aPageable, h_bPageable, d, numElem, "Pageable");
	profileCopy(h_aPinned, h_bPinned, d, numElem, "Pinned");

	hipHostFree(h_aPinned);
	hipHostFree(h_bPinned);
	free(h_aPageable);
	free(h_aPageable);
	hipFree(d);

	return 0;
}