#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
	if(result != hipSuccess)
		fprintf("CUDA Runtime Error: %s\n", hipGetErrorString(result));
	assert(result == hipSuccess);
#endif
	return result;
}

__global__
void arrAdd(float* arr, int offset)
{
	int idx = offset + threadIdx.x + blockIdx.x*blockDim.x;
	arr[idx] += 1.0;
}

float maxError(float *A, unsigned int n)
{
	float maxErr = 0.0;
	unsigned int i;
	for(i = 0; i < n; ++i)
		maxErr = max(maxErr, abs(A[i]-1.0));
	return maxErr;
}

int main(int argv, char** argv)
{
	const int numStrms = 4, blockSize = 256;
	const int n = 4 * 1024 * numStrms * blockSize;
	const int strmSize = n / numStrms;
	const int strmBytes = strmSize * sizeof(float);
	const int bytes = n * sizeof(float);

	//allocate pinned host memory and device memory
	float *A, *dA;
	checkCuda(hipHostMalloc((void**)&A, bytes));
	checkCuda(hipMalloc((void**)&dA, bytes));

	hipEvent_t stop, start;
	hipStream_t stream[numStrms];
	float milSecs;
	int i;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	for(i = 0; i < numStrms; i++)
		hipStreamCreate(&stream[i]);

	//base case for sequential transfer and execution
	memset(A, 0, bytes);
	hipEventRecord(start, 0);
	hipMemcpy(dA, A, bytes, hipMemcpyHostToDevice);
	arrAdd<<<n/blockSize, blockSize>>>(dA, 0);
	hipMemcpy(A, dA, bytes, hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milSecs, start, stop);
	printf("Time for sequential: %f", milSecs);
	printf("  max error: %f\n", maxError(A, n));

	//async v1: big loop
	memset(A, 0, bytes);
	hipEventRecord(start, 0);
	for(i = 0; i < numStrms; ++i)
	{
		offset = i * strmSize;
		hipMemcpyAsync(&dA[offset], &A[offset], strmBytes, hipMemcpyHostToDevice, &stream[i]);
		arrAdd<<<strmSize/blockSize, blockSize>>>(dA, offset);
		hipMemcpyAsync(&A[offset], &dA[offset], strmBytes, hipMemcpyDeviceToHost, &stream[i]);
	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milSecs, start, stop);
	printf("Time for async v1: %f", milSecs);
	printf("  max error: %f\n", maxError(A, n));

	//async v2: unrolled loops
	memset(A, 0, bytes);
	hipEventRecord(start, 0);
	for(i = 0; i < numStrms; ++i)
	{
		offset = i * strmSize;
		hipMemcpyAsync(&dA[offset], &A[offset], strmBytes, hipMemcpyHostToDevice, &stream[i]);
	}
	for(i = 0; i < numStrms; ++i)
	{
		offset = i * strmSize;
		arrAdd<<<strmSize/blockSize, blockSize>>>(dA, offset);
	}
	for(i = 0; i < numStrms; ++i)
	{
		offset = i * strmSize;
		hipMemcpyAsync(&A[offset], &dA[offset], strmSize, hipMemcpyDeviceToHost, &stream[i]);
	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milSecs, start, stop);
	printf("Time for async v2: %f\n", milSecs);
	printf("  max error: %f\n", maxError(A, n));

	//cleanup
	hipEventDestroy(start);
	hipEventDestroy(stop);
	for(i = 0; i < numStrms; ++i)
		hipStreamDestroy(stream[i]);
	hipHostFree(A);
	hipFree(dA);

	return 0;
}